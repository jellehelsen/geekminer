#include "hip/hip_runtime.h"
extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
#include "sph/sph_hamsi.h"
#include "sph/sph_shabal.h"

}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x11.h"
#include "x13/cuda_x13.h"

// to test gpu hash on a null buffer
#define NULLTEST 0

#include <stdio.h>
#include <memory.h>

static uint32_t *d_hash[MAX_GPUS];

// Geek CPU Hash
extern "C" void geekhash(void *output, const void *input)
{
	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
    #define hashA hash
    #define hashB hash+64

	memset(hash, 0, sizeof hash);

	// blake80-bmw512-echo512-shabal512-groestl512-cubehash512-keccak512-hamsi512-simd512

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_keccak512_context ctx_keccak;
	sph_cubehash512_context ctx_cubehash;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;
	sph_hamsi512_context ctx_hamsi;
	sph_shabal512_context ctx_shabal;


	sph_blake512_init(&ctx_blake);
	sph_blake512(&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, hashA);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512(&ctx_bmw, hashA, 64);
	sph_bmw512_close(&ctx_bmw, hashB);

	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, hashB, 64);
	sph_echo512_close(&ctx_echo, hashA);

	sph_shabal512_init(&ctx_shabal);
	sph_shabal512(&ctx_shabal, hashA, 64);
	sph_shabal512_close(&ctx_shabal, hashB);

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512(&ctx_groestl, hashB, 64);
	sph_groestl512_close(&ctx_groestl, hashA);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512(&ctx_cubehash, hashA, 64);
	sph_cubehash512_close(&ctx_cubehash, hashB);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512(&ctx_keccak, hashB, 64);
	sph_keccak512_close(&ctx_keccak, hashA);

	sph_hamsi512_init(&ctx_hamsi);
	sph_hamsi512(&ctx_hamsi, hashA, 64);
	sph_hamsi512_close(&ctx_hamsi, hashB);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, hashB, 64);
	sph_simd512_close(&ctx_simd, hashA);

	memcpy(output, hash, 32);
}

//#define _DEBUG
#define _DEBUG_PREFIX "geek"
#include "cuda_debug.cuh"

static bool init[MAX_GPUS] = { 0 };

extern "C" int scanhash_geek(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	int intensity = (device_sm[device_map[thr_id]] >= 500 && !is_windows()) ? 19 : 18;
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity); // 19=256*256*8;
	//if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
		ptarget[7] = 0x2;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		quark_blake512_cpu_init(thr_id, throughput);
		quark_bmw512_cpu_init(thr_id, throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id, throughput);
		quark_keccak512_cpu_init(thr_id, throughput);
		x11_cubehash512_cpu_init(thr_id, throughput);
		x11_simd512_cpu_init(thr_id, throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], (size_t) 64 * throughput), 0);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	quark_blake512_cpu_setBlock_80(thr_id, endiandata);
	cuda_check_cpu_setTarget(ptarget);

	do {
		int order = 0;

		// Hash with CUDA

		// blake80-bmw-echo-shabal-groestl-cubehash-keccak-hamsi-simd

		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id]); order++;
		quark_bmw512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_echo512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); order++;
		x14_shabal512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); order++;
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		quark_keccak512_cpu_hash_64(thr_id, throughput, NULL, d_hash[thr_id]); order++;
		x13_hamsi512_cpu_hash_64(thr_id, throughput, d_hash[thr_id]); order++;
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

#if NULLTEST
		uint32_t buf[8]; memset(buf, 0, sizeof buf);
		CUDA_SAFE_CALL(hipMemcpy(buf, d_hash[thr_id], sizeof buf, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		applog_hash(buf);
#endif

		*hashes_done = pdata[19] - first_nonce + throughput;

		work->nonces[0] = cuda_check_hash(thr_id, throughput, pdata[19], d_hash[thr_id]);
		if (work->nonces[0] != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t _ALIGN(64) vhash[8];
			be32enc(&endiandata[19], work->nonces[0]);
			geekhash(vhash, endiandata);

			if (vhash[7] <= Htarg && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				work->nonces[1] = cuda_check_hash_suppl(thr_id, throughput, pdata[19], d_hash[thr_id], 1);
				if (work->nonces[1] != 0) {
					be32enc(&endiandata[19], work->nonces[1]);
					geekhash(vhash, endiandata);
					bn_set_target_ratio(work, vhash, 1);
					work->valid_nonces++;
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1; // cursor
				}
				return work->valid_nonces;
			} else {
				gpu_increment_reject(thr_id);
				if (!opt_quiet)
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;
	return 0;
}

// cleanup
extern "C" void free_geek(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);

	quark_blake512_cpu_free(thr_id);
	quark_groestl512_cpu_free(thr_id);
	x11_simd512_cpu_free(thr_id);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
